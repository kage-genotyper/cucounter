#include <inttypes.h>
#include "common.h"
#include "util.h"

size_t get_free_cuda_memory()
{
  int device;
  hipGetDevice(&device);
  size_t free, total;
  hipMemGetInfo(&free, &total);
  return free;
}
